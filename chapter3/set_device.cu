/**
 * @file query_device.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2024-10-05
 * 
 * @copyright Copyright (c) 2024
 * 
 */

 #include "../common/book.h"

 int main ( void ) {
    hipDeviceProp_t prop;
    int dev; 
    
    HANDLE_ERROR( hipGetDeviceCount( &dev ));
    printf("ID of current CUDA device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));

    prop.major = 1;
    prop.minor = 3;

    HANDLE_ERROR( hipChooseDevice(&dev, &prop ));
    printf("ID of CUDA device closest to revision 1.3: %d\n", dev);
    HANDLE_ERROR( hipSetDevice(dev));
 }