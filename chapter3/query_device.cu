/**
 * @file query_device.cu
 * @author your name (you@domain.com)
 * @brief 
 * @version 0.1
 * @date 2024-10-05
 * 
 * @copyright Copyright (c) 2024
 * 
 */

 #include "../common/book.h"

 int main ( void ) {
    hipDeviceProp_t prop;

    int count; 
    HANDLE_ERROR( hipGetDeviceCount( &count ));
    for(int i=0; i < count; i++) {
        HANDLE_ERROR( hipGetDeviceProperties(&prop, i) );

        printf("  --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);

        printf("Device copy overlap:  ");
        if (prop.deviceOverlap) {
            printf("Enable\n");
        } else {
            printf("Disable\n");
        }

        printf("Kernel exection timeout :   ");
        if (prop.kernelExecTimeoutEnabled) {
            printf("Enable\n");
        } else {
            printf("Disable\n");
        }

        printf("   --- Memory Information for device %d --- \n", i);
        printf("Total global mem: %ld\n", prop.totalGlobalMem);
        printf("Total constant Mem: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment: %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d --- \n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("Registers per mp: %d\n", prop.regsPerBlock);
        printf("Threads in wrap: %d\n", prop.warpSize);
        printf("Max Threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max Thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("\n");
    }
 }